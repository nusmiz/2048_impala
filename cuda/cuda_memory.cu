
#include <hip/hip_runtime.h>
#include <cstddef>
#include <stdexcept>

namespace impala
{

void* allocatePinnedMemory(std::size_t size)
{
	void* ptr;
	auto ret = hipHostMalloc(&ptr, size, hipHostMallocDefault);
	if (ret != hipSuccess) {
		throw std::runtime_error("hipHostMalloc failed");
	}
	return ptr;
}

void freePinnedMemory(void* ptr)
{
	auto ret = hipHostFree(ptr);
	if (ret != hipSuccess) {
		throw std::runtime_error("hipHostFree failed");
	}
}

}  // namespace impala
